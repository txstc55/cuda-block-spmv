#include <algorithm>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hipsparse.h>
#include <fstream>
#include <iostream>
#include <map>
#include <vector>

#define ELEMENT_SIZE 3
#define DIMENSION 3
#define NUM_EXECUTION 100

#define CHECK_CUDA(call)                                                       \
  do {                                                                         \
    hipError_t error = call;                                                  \
    if (error != hipSuccess) {                                                \
      std::cerr << "CUDA Error: " << hipGetErrorString(error) << " at "       \
                << __FILE__ << ":" << __LINE__ << std::endl;                   \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

#define CHECK_CUSPARSE(call)                                                   \
  do {                                                                         \
    hipsparseStatus_t status = call;                                            \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      std::cerr << "cuSPARSE Error: " << status << " at " << __FILE__ << ":"   \
                << __LINE__ << std::endl;                                      \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

template <typename T>
std::vector<T> readVectorFromFile(const std::string &filename) {
  std::vector<T> vec;
  std::ifstream inFile(filename);
  T value;

  if (inFile.is_open()) {
    while (inFile >> value) {
      vec.push_back(value);
    }
  }

  inFile.close();
  return vec;
}

template <class F> __device__ __host__ inline F __m_min(F a, F b) {
  return a > b ? b : a;
}

__global__ void coo_spmv(const double *values, const int *elements,
                         const int dimension, const int inputSize,
                         const int numMatrices, const double *x, double *y) {
  // performs COO spmv y = Ax + y
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int matrixRowSize = dimension * inputSize;
  const int matrixSize = matrixRowSize * matrixRowSize;
  if (idx < numMatrices * matrixSize) {
    __shared__ int offset; // For example, matrix is 12 by 12, then with 32
                           // threads, then warp 1 will start with row 2 at
                           // element 8, this 4 is the offset
    int matrixID = idx / matrixSize; // which matrix are we working on
    int matrixRow = (idx % matrixSize) /
                    matrixRowSize; // which row of the matrix are we working on
    int matrixCol =
        (idx % matrixSize) %
        matrixRowSize; // which column of the matrix are we working on
    int smallBlockCol =
        matrixCol / dimension; // which column block are we working on
    int smallBlockColOffset =
        matrixCol %
        dimension; // inside this column block, which column are we working on
    int smallBlockRow = matrixRow / dimension;
    int smallBlockRowOffset = matrixRow % dimension;
    int colOffset = idx % matrixRowSize; // offset from one row

    double rdata =
        values[idx] *
        x[elements[matrixID * inputSize + smallBlockCol] * dimension +
          smallBlockColOffset];

    if (threadIdx.x == 0) {
      offset = (matrixRowSize - colOffset);
    }
    __syncthreads();

    int BRid = (threadIdx.x - offset + matrixRowSize) /
               matrixRowSize; // check if we are the first couple threads that
                              // are working on the first row in the warp
    int landidx = (threadIdx.x - offset) % matrixRowSize;
    if (BRid == 0) {
      landidx = threadIdx.x;
    }

    int warpId = threadIdx.x % 32;

    bool bBoundary = (landidx == 0) || (warpId == 0);

    unsigned int mask = __activemask();
    unsigned int mark = __ballot_sync(mask, bBoundary);
    mark = __brev(mark);
    unsigned int interval = min(__clz(mark << (warpId + 1)), 31 - warpId);

    for (int iter = 1; iter < matrixRowSize; iter <<= 1) {
      double tmp = __shfl_down_sync(mask, rdata, iter);
      if (interval >= iter)
        rdata += tmp;
    }

    // Only the thread that is the boundary (leader) writes the cpuResult
    if (bBoundary) {
      atomicAdd(&y[elements[matrixID * inputSize + smallBlockRow] * dimension +
                   smallBlockRowOffset],
                rdata);
    }
  }
}

__global__ void coo_spmv_segment(const double *values, const int *elements,
                                 const int dimension, const int inputSize,
                                 const int numMatrices, const double *x,
                                 double *y) {
  // performs COO spmv y = Ax + y
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  idx *= dimension;
  const int matrixRowSize = dimension * inputSize;
  const int matrixSize = matrixRowSize * matrixRowSize;
  if (idx < numMatrices * matrixSize) {
    __shared__ int offset; // For example, matrix is 12 by 12, then with 32
                           // threads, then warp 1 will start with row 2 at
                           // element 8, this 4 is the offset
    int matrixID = idx / matrixSize; // which matrix are we working on
    int matrixRow = (idx % matrixSize) /
                    matrixRowSize; // which row of the matrix are we working on
    int matrixCol =
        (idx % matrixSize) %
        matrixRowSize; // which column of the matrix are we working on
    int smallBlockCol =
        matrixCol / dimension; // which column block are we working on
    int smallBlockColOffset =
        matrixCol %
        dimension; // inside this column block, which column are we working on
    int smallBlockRow = matrixRow / dimension;
    int smallBlockRowOffset = matrixRow % dimension;
    int colOffset = idx % matrixRowSize; // offset from one row

    double rdata = 0.0;
    for (int i = 0; i < dimension; i++) {
      rdata += values[idx + i] *
               x[elements[matrixID * inputSize + smallBlockCol] * dimension +
                 smallBlockColOffset + i];
    }

    if (threadIdx.x == 0) {
      offset = (matrixRowSize - colOffset);
    }
    __syncthreads();

    int BRid = (threadIdx.x * dimension - offset + matrixRowSize) /
               matrixRowSize; // check if we are the first couple threads that
                              // are working on the first row in the warp
    int landidx = (threadIdx.x * dimension - offset) % matrixRowSize;
    if (BRid == 0 && threadIdx.x < matrixRowSize) {
      landidx = threadIdx.x;
    }

    int warpId = threadIdx.x % 32;

    bool bBoundary = (landidx == 0) || (warpId == 0);

    unsigned int mask = __activemask();
    unsigned int mark = __ballot_sync(mask, bBoundary);
    mark = __brev(mark);
    unsigned int interval = min(__clz(mark << (warpId + 1)), 31 - warpId);
    // printf("Thread %d, matrix %d, row %d, col %d, brid: %d, landidx: %d, "
    //        "interval %d, idx: %d, offset: %d\n",
    //        threadIdx.x, matrixID, matrixRow, matrixCol, BRid, landidx, interval,
    //        idx, offset);
    for (int iter = 1; iter < matrixRowSize; iter <<= 1) {
      double tmp = __shfl_down_sync(mask, rdata, iter);
      if (interval >= iter)
        rdata += tmp;
    }

    // Only the thread that is the boundary (leader) writes the cpuResult
    if (bBoundary) {
      atomicAdd(&y[elements[matrixID * inputSize + smallBlockRow] * dimension +
                   smallBlockRowOffset],
                rdata);
    //   printf("Boundary Thread %d, matrix %d, row %d, col %d, accumulatedRdata "
    //          "%lf, trow: %d, tcol: "
    //          "%d\n",
    //          threadIdx.x, matrixID, matrixRow, matrixCol, rdata,
    //          elements[matrixID * inputSize + smallBlockRow] * dimension +
    //              smallBlockRowOffset,
    //          elements[matrixID * inputSize + smallBlockCol] * dimension +
    //              smallBlockColOffset);
    }
  }
}

int main() {
  std::vector<int> elements = readVectorFromFile<int>("elements.txt");
  elements.resize((elements.size() / 4) * 3);

  std::vector<double> values =
      readVectorFromFile<double>("additionalHessianResults.txt");
  values.resize((values.size() / 144) * 81);

  int MATRIX_SIZE =
      *std::max_element(elements.begin(), elements.end()) * DIMENSION +
      DIMENSION;
  printf("Matrix size: %d\n", MATRIX_SIZE);

  // Expanded matrix calculations
  std::vector<int> expanded_rows(values.size());
  std::vector<int> expanded_cols(values.size());

  int count = 0;
  const int matrixSize = ELEMENT_SIZE * DIMENSION * ELEMENT_SIZE * DIMENSION;
  const int rowSize = ELEMENT_SIZE * DIMENSION;
  for (int idx = 0; idx < values.size(); idx++) {
    int matrixID = idx / matrixSize;
    int matrixRow = (idx % matrixSize) / rowSize;
    int matrixCol = (idx % matrixSize) % rowSize;
    int smallBlockRow = matrixRow / DIMENSION;
    int smallBlockCol = matrixCol / DIMENSION;
    int smallBlockRowOffset = matrixRow % DIMENSION;
    int smallBlockColOffset = matrixCol % DIMENSION;

    std::vector<int> localElements(ELEMENT_SIZE);
    for (int i = 0; i < ELEMENT_SIZE; i++) {
      localElements[i] = elements[matrixID * ELEMENT_SIZE + i];
    }

    int expandedRow =
        localElements[smallBlockRow] * DIMENSION + smallBlockRowOffset;
    int expandedCol =
        localElements[smallBlockCol] * DIMENSION + smallBlockColOffset;

    expanded_rows[count] = expandedRow;
    expanded_cols[count] = expandedCol;

    count++;
    if (expandedRow >= MATRIX_SIZE || expandedCol >= MATRIX_SIZE) {
      printf("Invalid expanded row/col: %d, %d\n", expandedRow, expandedCol);
      exit(EXIT_FAILURE);
    }
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;

  // read random x from file
  std::vector<double> h_x = readVectorFromFile<double>("fake_x.txt");
  // Allocate memory for vector x and y, and the random states
  double *d_x, *d_y_segment, *d_y;
  hiprandState *d_states;
  CHECK_CUDA(hipMalloc(&d_x, MATRIX_SIZE * sizeof(double)));
  CHECK_CUDA(hipMalloc(&d_y_segment, MATRIX_SIZE * sizeof(double)));
  CHECK_CUDA(hipMalloc(&d_y, MATRIX_SIZE * sizeof(double)));
  CHECK_CUDA(hipMemset(d_y_segment, 0, MATRIX_SIZE * sizeof(double)));
  CHECK_CUDA(hipMemset(d_y, 0, MATRIX_SIZE * sizeof(double)));
  CHECK_CUDA(hipMalloc(&d_states, MATRIX_SIZE * sizeof(hiprandState)));
  CHECK_CUDA(hipMemcpy(d_x, h_x.data(), MATRIX_SIZE * sizeof(double),
                        hipMemcpyHostToDevice));

  // get the true result from file
  std::vector<double> h_y(MATRIX_SIZE, 0.0);
  std::vector<double> trueResult = readVectorFromFile<double>("fake_y.txt");

  // let's do spmv
  std::vector<double> cpuResult(MATRIX_SIZE, 0.0);
  for (int i = 0; i < values.size(); i++) {
    int row = expanded_rows[i];
    int col = expanded_cols[i];
    cpuResult[row] += values[i] * h_x[col];
    // printf("i: %d, Row: %d, Col: %d, Value: %lf\n", i, row, col,
    //        values[i] * h_x[col]);
  }

//   // check the first 10 results of cpuResult
//   printf("Cpu first 10 result: \n");
//   for (int i = 0; i < 10; i++) {
//     printf("%lf ", cpuResult[i]);
//   }
//   printf("\n");

  // Allocate and copy COO format data
  int *d_expanded_rows, *d_expanded_cols, *d_elements;
  double *d_values;
  CHECK_CUDA(hipMalloc((void **)&d_expanded_rows,
                        expanded_rows.size() * sizeof(int)));
  CHECK_CUDA(hipMalloc((void **)&d_expanded_cols,
                        expanded_cols.size() * sizeof(int)));
  CHECK_CUDA(hipMalloc((void **)&d_values, values.size() * sizeof(double)));
  CHECK_CUDA(hipMalloc((void **)&d_elements, elements.size() * sizeof(int)));

  CHECK_CUDA(hipMemcpy(d_expanded_rows, expanded_rows.data(),
                        expanded_rows.size() * sizeof(int),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_expanded_cols, expanded_cols.data(),
                        expanded_cols.size() * sizeof(int),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_values, values.data(), values.size() * sizeof(double),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_elements, elements.data(),
                        elements.size() * sizeof(int), hipMemcpyHostToDevice));

  // ===========================================================
  // the segmented methodnvcc coo-block-spmv-3-by-3.cu -o main_3_3.out -O3
  // -gencode arch=compute_86,code=sm_86 -lcusparse
  // ===========================================================
  hipEventRecord(start);
  for (int i = 0; i < NUM_EXECUTION; i++) {
    coo_spmv_segment<<<(values.size() / DIMENSION + 31) / 32, 32>>>(d_values, d_elements, DIMENSION, ELEMENT_SIZE,
                                elements.size() / ELEMENT_SIZE, d_x,
                                d_y_segment);
  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Time averaged over %d executions for segmented spmv: %f\n",
         NUM_EXECUTION, milliseconds / NUM_EXECUTION);

  CHECK_CUDA(hipMemcpy(h_y.data(), d_y_segment, MATRIX_SIZE * sizeof(double),
                        hipMemcpyDeviceToHost));

//   printf("GPU first 10 result: \n");
//   for (int i = 0; i < 10; i++) {
//     printf("%lf ", h_y[i] / NUM_EXECUTION);
//   }
//   printf("\n");

  double error = 0.0;
  for (int i = 0; i < MATRIX_SIZE; i++) {
    error += abs(h_y[i] / NUM_EXECUTION - cpuResult[i]);
  }
  printf("Error for segmented with cpu: %f\n", error);

    // ===========================================================
    // the non segmented method
    // ===========================================================
    hipEventRecord(start);
    for (int i = 0; i < NUM_EXECUTION; i++) {
      coo_spmv<<<(values.size() + 31) / 32, 32>>>(
          d_values, d_elements, DIMENSION, ELEMENT_SIZE,
          elements.size() / ELEMENT_SIZE, d_x, d_y);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time averaged over %d executions for non segmented spmv: %f\n",
           NUM_EXECUTION, milliseconds / NUM_EXECUTION);

    CHECK_CUDA(hipMemcpy(h_y.data(), d_y, MATRIX_SIZE * sizeof(double),
                          hipMemcpyDeviceToHost));

    // printf("GPU first 10 result: \n");
    // for (int i = 0; i < 10; i++) {
    //   printf("%lf ", h_y[i] / NUM_EXECUTION);
    // }
    // printf("\n");

    error = 0.0;
    for (int i = 0; i < MATRIX_SIZE; i++) {
      error += abs(h_y[i] / NUM_EXECUTION - cpuResult[i]);
    }
    printf("Error for non segmented with cpu: %f\n", error);

    // Cleanup
    CHECK_CUDA(hipFree(d_values));
    CHECK_CUDA(hipFree(d_expanded_cols));
    CHECK_CUDA(hipFree(d_expanded_rows));
    CHECK_CUDA(hipFree(d_elements));
    CHECK_CUDA(hipFree(d_x));
    CHECK_CUDA(hipFree(d_y_segment));
    CHECK_CUDA(hipFree(d_states));

  return 0;
}
